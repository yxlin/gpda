#include "hip/hip_runtime.h"
#include <R.h>  // R Rprintf
//#include <iostream>        // includes, standard template & armadillo library
#include <armadillo>
#include "../inst/include/density.h"  
#include "../inst/include/reduce.h"

extern "C" void isp2(int *n, bool *out);

unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}


void isp2(int *n, bool *out) { *out = ((*n & (*n - 1)) == 0); }

/* -------------------------------------------------------------------------  
KDE operations 
------------------------------------------------------------------------- */
void summary(int *nsim, unsigned int *d_R, float *d_RT, float *out) {
  unsigned int maxThread = 256;
  unsigned int nThread = (*nsim < maxThread) ? nextPow2(*nsim) : maxThread;
  unsigned int nBlk    = ((*nsim) + nThread ) / nThread / 2;
  
  float *h_n1min_out, *h_n1max_out, *h_sum_out, *h_sqsum_out;
  float *d_n1min_out, *d_n1max_out, *d_sum_out, *d_sqsum_out;
  unsigned int *h_count_out, *h_nsim;
  unsigned int *d_count_out, *d_nsim;
  
  size_t dBlkfSize = nBlk * sizeof(float) * 2;
  size_t blkfSize  = nBlk * sizeof(float);
  size_t dBlkuSize = nBlk * sizeof(unsigned int) * 2;
  size_t uSize     = 1 * sizeof(unsigned int);
  
  h_nsim      = (unsigned int *)malloc(uSize);
  h_n1min_out = (float *)malloc(blkfSize);
  h_n1max_out = (float *)malloc(blkfSize);
  h_sum_out   = (float *)malloc(blkfSize);
  h_sqsum_out = (float *)malloc(dBlkfSize);
  h_count_out = (unsigned int *)malloc(dBlkuSize);
  // must reset h_count_out back to 0
  for(int i=0; i<2*nBlk; i++) { h_count_out[i] = 0; } 
  *h_nsim = (unsigned int)*nsim;
  
  hipMalloc((void**) &d_nsim,      uSize);
  hipMalloc((void**) &d_n1min_out, blkfSize);
  hipMalloc((void**) &d_n1max_out, blkfSize);
  hipMalloc((void**) &d_sum_out,   blkfSize);
  hipMalloc((void**) &d_sqsum_out, dBlkfSize);
  hipMalloc((void**) &d_count_out, dBlkuSize);
  
  hipMemcpy(d_nsim,      h_nsim,  uSize,  hipMemcpyHostToDevice);
  hipMemcpy(d_count_out, h_count_out, dBlkuSize, hipMemcpyHostToDevice);
  
  // must be first min and then max
  count_kernel<<<2*nBlk, nThread>>>(d_nsim, d_R, d_count_out); hipFree(d_R);
  n1min_kernel<<<nBlk, nThread>>>(d_RT, d_n1min_out); 
  n1max_kernel<<<nBlk, nThread>>>(d_RT, d_n1max_out);
  sum_kernel<<<nBlk, nThread>>>(d_RT,   d_sum_out);
  squareSum_kernel<<<2*nBlk, nThread>>>(d_nsim, d_RT, d_sqsum_out);
  
  hipMemcpy(h_n1min_out, d_n1min_out, blkfSize,  hipMemcpyDeviceToHost); hipFree(d_n1min_out);
  hipMemcpy(h_n1max_out, d_n1max_out, blkfSize,  hipMemcpyDeviceToHost); hipFree(d_n1max_out);
  hipMemcpy(h_sum_out,   d_sum_out,   blkfSize,  hipMemcpyDeviceToHost); hipFree(d_sum_out);
  hipMemcpy(h_sqsum_out, d_sqsum_out, dBlkfSize, hipMemcpyDeviceToHost); hipFree(d_sqsum_out);
  hipMemcpy(h_count_out, d_count_out, dBlkuSize, hipMemcpyDeviceToHost); hipFree(d_count_out);
  
  arma::vec min_tmp(nBlk); arma::vec max_tmp(nBlk);
  float sum = 0, sqsum = 0;
  for (int i=0; i<2*nBlk; i++) {
    sqsum += h_sqsum_out[i];
    if ( i < nBlk ) {
      min_tmp[i] = (double)h_n1min_out[i];
      max_tmp[i] = (double)h_n1max_out[i];
      sum += h_sum_out[i];
    }
  }
  
  free(h_sqsum_out); free(h_n1min_out); free(h_n1max_out); free(h_sum_out);
  out[0] = min_tmp.min();
  out[1] = max_tmp.max();
  out[2] = std::sqrt( (sqsum - (sum*sum) / h_count_out[0]) / (h_count_out[0] - 1) );
  out[3] = h_count_out[0]; free(h_count_out);
  
  // Rprintf("RT0 [minimum maximum]: %.2f %.2f\n", min_tmp.min(), max_tmp.max());
  // Rprintf("RT0 [sum sqsum]: %.2f %.2f\n", sum, sqsum);
  // Rprintf("RT0 [nsRT0 sd]: %f %f\n", out[3], out[2]);
  free(h_nsim); hipFree(d_nsim);
}


void histc(int *nsim, int ngrid, float *h_binedge, float *d_RT, unsigned int *h_hist)
{
  size_t ngrid_plus1fSize = (ngrid + 1) * sizeof(float);
  size_t ngriduSize = ngrid * sizeof(unsigned int);
  
  float *d_binedge;
  unsigned int *d_hist;
  unsigned int *h_nsim, *d_nsim;
  h_nsim  = (unsigned int *)malloc(sizeof(unsigned int) * 1);
  *h_nsim = (unsigned int)*nsim;
  hipMalloc((void**) &d_nsim, sizeof(unsigned int) * 1);
  hipMemcpy(d_nsim,   h_nsim, sizeof(unsigned int) * 1,  hipMemcpyHostToDevice);
  hipMalloc((void**) &d_binedge, ngrid_plus1fSize); // 1025
  hipMalloc((void**) &d_hist,    ngriduSize);       // 1024
  hipMemcpy(d_binedge, h_binedge, ngrid_plus1fSize, hipMemcpyHostToDevice); free(h_binedge);
  hipMemcpy(d_hist,    h_hist,    ngriduSize,       hipMemcpyHostToDevice);
  histc_kernel<<<*nsim/ngrid, ngrid>>>(d_binedge, d_RT, d_nsim, d_hist);
  hipFree(d_RT); hipFree(d_binedge); hipFree(d_nsim);
  
  hipMemcpy(h_hist, d_hist, ngriduSize, hipMemcpyDeviceToHost); 
  hipFree(d_hist); free(h_nsim); 
}
